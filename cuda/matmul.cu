#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define TILE_SIZE 16

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA error: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void matrixMulKernel(float *A, float *B, float *C, int N) {
    __shared__ float sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedB[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;
    float sum = 0.0f;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        if (row < N && t * TILE_SIZE + tx < N)
            sharedA[ty][tx] = A[row * N + t * TILE_SIZE + tx];
        else
            sharedA[ty][tx] = 0.0f;

        if (col < N && t * TILE_SIZE + ty < N)
            sharedB[ty][tx] = B[(t * TILE_SIZE + ty) * N + col];
        else
            sharedB[ty][tx] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++) {
            sum += sharedA[ty][k] * sharedB[k][tx];
        }

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

void matrixMultiply(float *h_A, float *h_B, float *h_C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matrixMulKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();
    gpuAssert(hipGetLastError(), __FILE__, __LINE__);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 64; // Matrix size
    size_t size = N * N * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 10;
        h_B[i] = rand() % 10;
    }

    matrixMultiply(h_A, h_B, h_C, N);

    printf("Result matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%0.1f ", h_C[i * N + j]);
        }
        printf("\n");
    }

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
